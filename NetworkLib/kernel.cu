#include "hip/hip_runtime.h"
﻿#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#include "kernel.h"

__global__ void krelu(float* outputs, float* activations, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        activations[idx] = fmaxf(0.0f, outputs[idx]);
    }
}

void NetworkLib::Gpu::Kernel::relu(hipStream_t stream, float* outputs, float* activations, int size) {
    int threadsPerBlock = 64;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    krelu <<<blocksPerGrid, threadsPerBlock, 0, stream >>>(outputs, activations, size);
}
