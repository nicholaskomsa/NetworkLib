#include "hip/hip_runtime.h"
﻿#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#include "kernel.h"

namespace Kernel = NetworkLib::Gpu::Kernel;

__global__ void cuRelu(const float* outputs, float* reluActivations, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) 
        reluActivations[idx] = fmaxf(0.0f, outputs[idx]);
}
__device__ float kReluPrime(float value) {
    return value < 0.0f ? 0.0f : 1.0f;
}

__global__ void cuApplyReluPrime(const float* reluActivations, float* reluPrimes, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
        reluPrimes[idx] *= kReluPrime(reluActivations[idx]);
}

__global__ void cuSoftmax1024(const float* outputs, float* softmaxActivations, int size) {
    extern __shared__ float shared_exp[];

    int tid = threadIdx.x;

    if (tid < size) 
        shared_exp[tid] = expf(outputs[tid]);
    
    __syncthreads();

    // Block-local reduction
    float sum = 0.0f;
    if (tid == 0) {
        for (int i = 0; i < size; ++i) {
            sum += shared_exp[i];
        }
        shared_exp[0] = sum;
    }
    __syncthreads();

    if (tid < size) 
        softmaxActivations[tid] = shared_exp[tid] / shared_exp[0];
}
__global__ void cuDiff(const float* desired, const float* sought, float* primes, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) 
        primes[i] = sought[i] - desired[i];
}
__global__ void cuBatchedCopy(const float* src, float* dst, int size, int batchSize) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = blockIdx.y;

    if (row < size && col < batchSize) {
        int i = row + col * size; // column-major offset
        dst[i] = src[i];
    }
}

__global__ void cuUpdateWeights(float* weights, const float* primes, const float* seen, int r, int c, float learnRate) {
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Column index
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Row index

    if (col < c && row < r) {
        int index_col_major = row + col * r;

        weights[index_col_major] -= primes[row] * seen[col] * learnRate;
    }
}
__global__ void cuBroadcastVectorToColumns(const float* src, float* dst, int size, int batchSize) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = blockIdx.y;

    if (row < size && col < batchSize) {
        dst[row + col * size] = src[row];  // column-major offset
    }
}
__global__ void cuBroadcastVectorToColumnsAdd(const float* src, float* dst, int size, int batchSize) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = blockIdx.y;

    if (row < size && col < batchSize) {
        dst[row + col * size] += src[row];  // column-major offset
    }
}


void Kernel::relu(hipStream_t stream, const float* outputs, float* reluActivations, int size) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    cuRelu <<<blocksPerGrid, threadsPerBlock, 0, stream >>>(outputs, reluActivations, size);
}
void Kernel::applyReluPrime(hipStream_t stream, const float* reluActivations, float* primes, int size) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    cuApplyReluPrime <<<blocksPerGrid, threadsPerBlock, 0, stream >>> (reluActivations, primes, size);
}
void Kernel::softmax(hipStream_t stream, const float* outputs, float* softmaxActivations, int size) {
    cuSoftmax1024 <<<1, size, size * sizeof(float), stream >>>(outputs, softmaxActivations, size);
}
void Kernel::diff(hipStream_t stream, const float* desired, const float* sought, float* primes, int size) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    cuDiff <<<1, size, 0, stream >>> (desired, sought, primes, size);
}
void Kernel::updateWeights(hipStream_t stream, float* weights, const float* primes, const float* seen, int rows, int cols, float learnRate) {

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((cols + 32 - 1) / 32, (rows + 32 - 1) / 32);
    cuUpdateWeights<<<numBlocks, threadsPerBlock, 0, stream>>>(weights, primes, seen, rows, cols, learnRate);
}
void Kernel::batchedCopy(hipStream_t stream, const float* src, float* dst, int size, int batchSize) {
    int threadsPerBlock = 256;
    int blocksPerRow = (size + threadsPerBlock - 1) / threadsPerBlock;

    dim3 grid(blocksPerRow, batchSize);
    dim3 block(threadsPerBlock);

    cuBatchedCopy<<<grid, block, 0, stream >>>(src, dst, size, batchSize);
}
void Kernel::batchedBroadcast(hipStream_t stream, const float* src, float* dst, int size, int batchSize) {
    int threadsPerBlock = 256;
    int blocksPerRow = (size + threadsPerBlock - 1) / threadsPerBlock;

    dim3 grid(blocksPerRow, batchSize);
    dim3 block(threadsPerBlock);

    cuBroadcastVectorToColumns<<<grid, block, 0, stream>>>(src, dst, size, batchSize);
}
void Kernel::batchedBroadcastAdd(hipStream_t stream, const float* src, float* dst, int size, int batchSize) {
    int threadsPerBlock = 256;
    int blocksPerRow = (size + threadsPerBlock - 1) / threadsPerBlock;

    dim3 grid(blocksPerRow, batchSize);
    dim3 block(threadsPerBlock);

    cuBroadcastVectorToColumnsAdd << <grid, block, 0, stream >> > (src, dst, size, batchSize);
}
