#include "hip/hip_runtime.h"
﻿#include <stdio.h>

#include "hip/hip_runtime.h"
#include ""

#include "kernel.h"

namespace Kernel = NetworkLib::Gpu::Kernel;

__global__ void cuRelu(const float* outputs, float* reluActivations, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) 
        reluActivations[idx] = fmaxf(0.0f, outputs[idx]);
}
__device__ float kReluPrime(float value) {
    return value < 0.0f ? 0.0f : 1.0f;
}

__global__ void cuApplyReluPrime(const float* reluActivations, float* reluPrimes, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
        reluPrimes[idx] *= kReluPrime(reluActivations[idx]);
}

__global__ void cuSoftmax1024(const float* outputs, float* softmaxActivations, int size) {
    extern __shared__ float shared_exp[];

    int tid = threadIdx.x;

    if (tid < size) 
        shared_exp[tid] = expf(outputs[tid]);
    
    __syncthreads();

    // Block-local reduction
    float sum = 0.0f;
    if (tid == 0) {
        for (int i = 0; i < size; ++i) {
            sum += shared_exp[i];
        }
        shared_exp[0] = sum;
    }
    __syncthreads();

    if (tid < size) 
        softmaxActivations[tid] = shared_exp[tid] / shared_exp[0];
}
__global__ void cuDiff(const float* desired, const float* sought, float* primes, int size) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < size) 
        primes[i] = sought[i] - desired[i];
}
__global__ void cuBatchedCopy(const float* src, float* dst, int size, int batchSize) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = blockIdx.y;

    if (row < size && col < batchSize) {
        int i = row + col * size; // column-major offset
        dst[i] = src[i];
    }
}

__global__ void cuUpdateWeights(float* weights, const float* primes, const float* seen, int r, int c, float learnRate) {
    int col = blockIdx.x * blockDim.x + threadIdx.x; // Column index
    int row = blockIdx.y * blockDim.y + threadIdx.y; // Row index

    if (col < c && row < r) {
        int index_col_major = row + col * r;

        weights[index_col_major] -= primes[row] * seen[col] * learnRate;
    }
}
__global__ void cuBatchedUpdateWeights(float* weights, const float* primes, const float* seen, int rows, int cols, int batchSize, float learnRate) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int b = blockIdx.z;  // Each block handles one sample

    if (col < cols && row < rows && b < batchSize) {
        float prime_val = primes[b * rows + row];   // primes[row, b]
        float seen_val = seen[b * cols + col];     // seen[col, b]

        int index_col_major = row + col * rows;

        atomicAdd(&weights[index_col_major], -learnRate * prime_val * seen_val);
    }

}
__global__ void cuBroadcastVectorToColumns(const float* src, float* dst, int size, int batchSize) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = blockIdx.y;

    if (row < size && col < batchSize) {
        dst[row + col * size] = src[row];  // column-major offset
    }
}
__global__ void cuBroadcastVectorToColumnsAdd(const float* src, float* dst, int size, int batchSize) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = blockIdx.y;

    if (row < size && col < batchSize) {
        dst[row + col * size] += src[row];  // column-major offset
    }
}
__global__ void cuMse(const float* sought, const float* desired, float* result, int size, int batchSize) {
    extern __shared__ float partialSum[]; // shared memory per block

    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = blockIdx.y;
    int tid = threadIdx.x;

    float localSum = 0.0f;

    if (row < size && col < batchSize) {
        int i = row + col * size; // column-major offset
        float diff = sought[i] - desired[i];
        localSum = diff * diff;
    }

    partialSum[tid] = localSum;
    __syncthreads();

    // Block-level reduction
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            partialSum[tid] += partialSum[tid + stride];
        }
        __syncthreads();
    }

    // Accumulate per-batch sum into global result
    if (tid == 0) {
        float batchMse = partialSum[0] / size / batchSize;
        atomicAdd(result, batchMse);
    }

}

void Kernel::mse(hipStream_t stream, const float* sought, const float* desired, float* result, int size, int batchSize) {
    int threadsPerBlock = 256;
    int blocksPerBatch = (size + threadsPerBlock - 1) / threadsPerBlock;
    dim3 grid(blocksPerBatch, batchSize);   // one block per batch row
    dim3 block(threadsPerBlock);
    size_t sharedMemSize = threadsPerBlock * sizeof(float);

    cuMse<<<grid, block, sharedMemSize, stream>>>(sought, desired, result, size, batchSize);
}

void Kernel::relu(hipStream_t stream, const float* outputs, float* reluActivations, int size) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    cuRelu <<<blocksPerGrid, threadsPerBlock, 0, stream >>>(outputs, reluActivations, size);
}
void Kernel::applyReluPrime(hipStream_t stream, const float* reluActivations, float* primes, int size) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    cuApplyReluPrime <<<blocksPerGrid, threadsPerBlock, 0, stream >>> (reluActivations, primes, size);
}
void Kernel::softmax(hipStream_t stream, const float* outputs, float* softmaxActivations, int size) {
    cuSoftmax1024 <<<1, size, size * sizeof(float), stream >>>(outputs, softmaxActivations, size);
}
void Kernel::diff(hipStream_t stream, const float* desired, const float* sought, float* primes, int size) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    cuDiff <<<1, size, 0, stream >>> (desired, sought, primes, size);
}
void Kernel::updateWeights(hipStream_t stream, float* weights, const float* primes, const float* seen, int rows, int cols, float learnRate) {

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((cols + 32 - 1) / 32, (rows + 32 - 1) / 32);
    cuUpdateWeights<<<numBlocks, threadsPerBlock, 0, stream>>>(weights, primes, seen, rows, cols, learnRate);
}
void Kernel::batchedCopy(hipStream_t stream, const float* src, float* dst, int size, int batchSize) {
    int threadsPerBlock = 256;
    int blocksPerRow = (size + threadsPerBlock - 1) / threadsPerBlock;

    dim3 grid(blocksPerRow, batchSize);
    dim3 block(threadsPerBlock);

    cuBatchedCopy<<<grid, block, 0, stream >>>(src, dst, size, batchSize);
}
void Kernel::batchedBroadcast(hipStream_t stream, const float* src, float* dst, int size, int batchSize) {
    int threadsPerBlock = 256;
    int blocksPerRow = (size + threadsPerBlock - 1) / threadsPerBlock;

    dim3 grid(blocksPerRow, batchSize);
    dim3 block(threadsPerBlock);

    cuBroadcastVectorToColumns<<<grid, block, 0, stream>>>(src, dst, size, batchSize);
}
void Kernel::batchedBroadcastAdd(hipStream_t stream, const float* src, float* dst, int size, int batchSize) {
    int threadsPerBlock = 256;
    int blocksPerRow = (size + threadsPerBlock - 1) / threadsPerBlock;

    dim3 grid(blocksPerRow, batchSize);
    dim3 block(threadsPerBlock);

    cuBroadcastVectorToColumnsAdd << <grid, block, 0, stream >> > (src, dst, size, batchSize);
}
void Kernel::batchedUpdateWeights(hipStream_t stream, float* weights, const float* primes, const float* seen, int rows, int cols, int batchSize, float learnRate) {
    dim3 blockDim(16, 16, 1);  // Threads per block
    dim3 gridDim((cols + 15) / 16, (rows + 15) / 16, batchSize);  // One block per batch
    cuBatchedUpdateWeights << <gridDim, blockDim, 0, stream >> >(weights, primes, seen, rows, cols, batchSize, learnRate);

}