#include "hip/hip_runtime.h"
#include <stdio.h>

#include <cfloat>

#include "hip/hip_runtime.h"
#include ""

#include "kernel.h"

namespace Kernel = NetworkLib::Gpu::Kernel;

__global__ void cuConv1(float* seen, float* weights, float* primes, int primesSize, int kernelWidth, int kernelDepth, int kernel) {

    int p = blockIdx.x * blockDim.x + threadIdx.x; // output index

    if (p < primesSize) {

        float sum = 0.0f;
        for (int w = 0; w < kernelWidth; ++w)
            sum += weights[kernel * kernelWidth + w] * seen[p + w];

        primes[kernel * primesSize + p] = sum;
    }
}
void Kernel::conv1(hipStream_t stream, float* weights, float* primes, float* seen, int primesSize, int kernelWidth, int kernelDepth, int kernel) {
    int kprimesSize = std::max(1, primesSize / kernelDepth);
    int tx = std::min(32, kprimesSize);       // threads per block in x ? output positions

    dim3 threadsPerBlock(tx);
    dim3 numBlocks((kprimesSize + tx - 1) / tx);

    cuConv1 << <numBlocks, threadsPerBlock, 0, stream >> > (seen, weights, primes, kprimesSize, kernelWidth, kernelDepth, kernel);

}
__global__ void cuBatchedConv1(float* seen, float* weights, float* primes, int primesSize, int kernelWidth, int kernelDepth, int batchSize) {
    /*
    int p = blockIdx.x * blockDim.x + threadIdx.x; // output index
    int k = blockIdx.y * blockDim.y + threadIdx.y; // kernel index
    int b = blockIdx.z * blockDim.z + threadIdx.z;

    if (k < kernelDepth && p < primesSize && b < batchSize) {

        std::size_t seenBatchOffset = (primesSize + kernelWidth - 1) * b + p;
        std::size_t primesBatchOffset = primesSize * kernelDepth * b + k * primesSize + p;

        for (int w = 0; w < kernelWidth; ++w)
            atomicAdd(&primes[primesBatchOffset], weights[k * kernelWidth + w] * seen[seenBatchOffset + w]);
    }
    */
}
void Kernel::batchedConv1(hipStream_t stream, float* weights, float* output, float* seen, int primesSize, int kernelSize, int kernelDepth, int batchSize) {
    int kprimesSize = std::max(1, primesSize / kernelDepth);
    int tx = std::min(32, kprimesSize);       // threads per block in x ? output positions
    int ty = std::min(32, kernelDepth);      // threads per block in y ? kernel depth
    int tz = 0;

    //x* y* z <= 1024
    if (tx * ty * batchSize < 1024)
        tz = batchSize;
    else
        tz = std::min(batchSize, 1024 / (tx * ty));

    dim3 threadsPerBlock(tx, ty, tz);
    dim3 numBlocks(
        (kprimesSize + tx - 1) / tx,
        (kernelDepth + ty - 1) / ty,
        (batchSize + tz - 1) / tz
    );

    cuBatchedConv1 << <numBlocks, threadsPerBlock, 0, stream >> > (seen, weights, output, kprimesSize, kernelSize, kernelDepth, batchSize);
}
__global__ void cuConv1VecMulVec(float* weights, float* errors, float* primes, int kernelWidth, int kPrimesSize, int kernelDepth, int kernel) {

    int p = blockIdx.x * blockDim.x + threadIdx.x;

    if (p < kPrimesSize) {

        int idx = kPrimesSize * kernel + p, wOffset = kernelWidth * kernel;

        float sum = 0.0f;
        for (int w = 0; w < kernelWidth; ++w)
            sum += weights[wOffset + w] * errors[idx];

        primes[idx] = sum;
    }
}

void Kernel::conv1VecMulVec(hipStream_t stream, float* weights, float* errors, float* primes, int primesSize, int kernelWidth, int kernelDepth, int kernel) {

    int kprimesSize = std::max(1, primesSize / kernelDepth);
    int tx = std::min(32, kprimesSize);       // threads per block in x ? output positions

    dim3 threadsPerBlock(tx);
    dim3 numBlocks((kprimesSize + tx - 1) / tx);

    cuConv1VecMulVec << <numBlocks, threadsPerBlock, 0, stream >> > (weights, errors, primes, kernelWidth, kprimesSize, kernelDepth, kernel);

}

__global__ void cuBatchedConv1VecMulVec(float* weights, float* errors, float* primes, int kernelSize, int primesSize, int kernelDepth, int batchSize) {

    int p = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;
    int b = blockIdx.z * blockDim.z + threadIdx.z;

    if (p < primesSize && k < kernelDepth && b < batchSize) {

        int batchOffset = primesSize * kernelDepth * b;
        int idx = batchOffset + primesSize * k + p, wOffset = kernelSize * k;

        float e = errors[idx];

        for (int w = 0; w < kernelSize; ++w)
            atomicAdd(&primes[idx], weights[wOffset + w] * e);
    }
}
void Kernel::batchedConv1VecMulVec(hipStream_t stream, float* weights, float* errors, float* primes, int primesSize, int kernelWidth, int kernelDepth, int batchSize) {

    int kprimesSize = std::max(1, primesSize / kernelDepth);
    int tx = std::min(32, kprimesSize);       // output positions
    int ty = std::min(32, kernelDepth);       // kernel depth
    int tz = 0;

    //x* y* z <= 1024
    if (tx * ty * batchSize < 1024)
        tz = batchSize;
    else
        tz = std::min(batchSize, 1024 / (tx * ty));

    dim3 threadsPerBlock(tx, ty, tz);
    dim3 numBlocks(
        (kprimesSize + tx - 1) / tx,
        (kernelDepth + ty - 1) / ty,
        (batchSize + tz - 1) / tz
    );

    cuBatchedConv1VecMulVec << <numBlocks, threadsPerBlock, 0, stream >> > (weights, errors, primes, kernelWidth, kprimesSize, kernelDepth, batchSize);

}
__global__ void cuConv1UpdateKernel(float* seen, float* weights, float* primes, int kPrimesSize, int kernelWidth, int kernel, float learnRate) {

    int p = blockIdx.x * blockDim.x + threadIdx.x; // kprimes index

    if (p < kPrimesSize) {

        int primesOffset = kPrimesSize * kernel;

        int kernelHeight = 1;

        float prime_val = primes[primesOffset + p] * -learnRate / kPrimesSize;

        for (int x = 0; x < kernelWidth; ++x) {
            for (int y = 0; y < kernelHeight; ++y) {

                int index_col_major = y + kernelHeight * (x + kernelWidth * kernel);
                atomicAdd(&weights[index_col_major], prime_val * seen[p + x]); //seen[p * kernelWidth + x] 2d seen
            }
        }
    }
}
void Kernel::conv1UpdateKernel(hipStream_t stream, float* weights, float* primes, float* seen, int primesSize, int kernelSize, int kernelDepth, int kernel, float learnRate) {

    int kprimesSize = std::max(1, primesSize / kernelDepth);
    int tx = std::min(32, kprimesSize);       // threads per block in x ? output positions

    dim3 threadsPerBlock(tx);
    dim3 numBlocks((kprimesSize + tx - 1) / tx);

    cuConv1UpdateKernel<<<numBlocks, threadsPerBlock, 0, stream>>>(seen, weights, primes, kprimesSize, kernelSize, kernel, learnRate);
}

__global__ void cuBatchedConv1UpdateKernel(float* seen, float* weights, float* primes, int kPrimesSize, int kernelWidth, int kernelDepth, int batchSize, float learnRate) {

    /*
    *         int index_col_major = row + col * rows;
    */
    /*no work
    int p = blockIdx.x * blockDim.x + threadIdx.x; // output index
    int k = blockIdx.y * blockDim.y + threadIdx.y; // kernel index
    int b = blockIdx.z * blockDim.z + threadIdx.z;

    if (k < kernelDepth && p < kPrimesSize && b < batchSize) {

        std::size_t inputSize = kPrimesSize + kernelWidth - 1;
        std::size_t seenBatchOffset = inputSize * b + p;
        std::size_t primesBatchOffset = kPrimesSize * kernelDepth * b + k * kPrimesSize + p;

        float prime_val = -learnRate * primes[primesBatchOffset] / kPrimesSize;
        int kernelOffset = k * kernelWidth;

        for (int w = 0; w < kernelWidth; ++w)
            atomicAdd(&weights[kernelOffset + w], prime_val * seen[seenBatchOffset+w]);
    }
    */
}
void Kernel::batchedConv1UpdateKernel(hipStream_t stream, float* weights, float* primes, float* seen, int primesSize, int kernelSize, int kernelDepth, int batchSize, float learnRate) {

    int kprimesSize = std::max(1, primesSize / kernelDepth);
    int tx = std::min(32, kprimesSize);       // threads per block in x ? output positions
    int ty = std::min(32, kernelDepth);      // threads per block in y ? kernel depth
    int tz = 0;

    //x* y* z <= 1024
    if (tx * ty * batchSize < 1024)
        tz = batchSize;
    else
        tz = std::min(batchSize, 1024 / (tx * ty));

    dim3 threadsPerBlock(tx, ty, tz);
    dim3 numBlocks(
        (kprimesSize + tx - 1) / tx,
        (kernelDepth + ty - 1) / ty,
        (batchSize + tz - 1) / tz
    );
    cuBatchedConv1UpdateKernel<<<numBlocks, threadsPerBlock, 0, stream>>>(seen, weights, primes, primesSize, kernelSize, kernelDepth, batchSize, learnRate);
}